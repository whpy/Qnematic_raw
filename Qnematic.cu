#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <string>
#include <cmath>
#include <chrono>
#include <random>

#include <cuComplexBinOp.cuh>
#include <cudaErr.h>
#include <QActFlowDef.cuh>

#include <Mesh.cuh>
#include <Qfunctions.cuh>
#include <QFldfuncs.cuh>
// #include <Qderiv.cuh>

// #define M_PI 3.141592653589

using namespace std;
// struct to faciliate
// typedef struct Field{
//     Qreal *phys;
//     Qcomp *spec;
//     Field(Qreal *pphys, Qcomp* pspec):phys(pphys), spec(pspec){}
// } Field;
// inline void waveinit(Qreal* kx, Qreal* ky, Qreal* k_squared, int Nxh, int Ny, Qreal Lx, Qreal Ly);
// funtions
// __global__ 
// void coeff(Qreal *f, int Nx, int Ny, int BSZ){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = j*Nx + i;
//     if(i<Nx && j<Ny){
//         f[index] = f[index]/(Nx*Ny);
//     }
// }
// void FwdTrans(Qcomp* ft, Qreal* f, Mesh* mesh){
//     hipMemcpy(mesh->phys, f, mesh->physsize, hipMemcpyDeviceToDevice);
//     cufft_error_func( hipfftExecD2Z(mesh->transf, mesh->phys, ft));
// }

// void BwdTrans( Qreal* f, Qcomp* ft, Mesh* mesh){
//     hipMemcpy(mesh->spec, ft, mesh->specsize, hipMemcpyDeviceToDevice);
//     cufft_error_func( hipfftExecZ2D(mesh->inv_transf, mesh->spec, f));
//     coeff<<<mesh->dimGridp,mesh->dimBlockp>>>(f, mesh->Nx, mesh->Ny, mesh->BSZ);
// }

// inline void field_visual(Qreal *f, string name, int Nx, int Ny){
//     ofstream fval;
//     string fname = name;
//     fval.open(fname);
//     for (int j=0; j<Ny; j++){
//         for (int i=0; i<Nx; i++){
//             int index = j*Nx + i;
//             fval << f[index] << ",";
//         }
//         fval << endl;
//     }
//     fval.close();
// }

// inline void waveinit(Qreal* kx, Qreal* ky, Qreal* k_squared, int Nxh, int Ny, Qreal Lx, Qreal Ly){
//     for (int i = 0; i < Nxh; i++){
//         for (int j = 0; j < Ny; j++){
//             int index = i + j*Nxh;
            
//             if (j<Ny/2+1){
//                 ky[index] = 2*M_PI/Ly * j;
//             }
//             else{
//                 ky[index] = 2*M_PI/Ly * (j-Ny);
//             }
//             kx[index] = 2*M_PI/Lx * i;
//             k_squared[index] = kx[index]*kx[index] + ky[index]*ky[index]; 
//         }
//     }
// }

// inline void winit(Qreal* w, int Nx, int Ny, Qreal dx, Qreal dy){
//     for (int j = 0; j < Ny; j++){
//         for (int i = 0; i < Nx; i++){
//             int index = i + j*Nx;
//             Qreal x = i*dx;
//             Qreal y = j*dy;
//             w[index] = -1*sin(x+y);
//         }
//     }
// }

// inline void r1init(Qreal* w, int Nx, int Ny, Qreal dx, Qreal dy){
//     for (int j = 0; j < Ny; j++){
//         for (int i = 0; i < Nx; i++){
//             int index = i + j*Nx;
//             Qreal x = i*dx;
//             Qreal y = j*dy;
//             w[index] = -25*sin(3*x+4*y);
//         }
//     }
// }

// inline void r2init(Qreal* w, int Nx, int Ny, Qreal dx, Qreal dy){
//     for (int j = 0; j < Ny; j++){
//         for (int i = 0; i < Nx; i++){
//             int index = i + j*Nx;
//             Qreal x = i*dx;
//             Qreal y = j*dy;
//             w[index] = -25*sin(3*x+4*y);
//         }
//     }
// }

// inline void print_spec(Qcomp* f, int Nxh, int Ny){
//     for(int j = 0; j < Ny; j++){
//         for (int i = 0; i < Nxh; i++){
//             int index = i + j*Nxh;
//             printf("(%.2f, %.2f)  ", f[index].x, f[index].y);
//         }
//         cout << endl;
//     }
//     cout << endl;
// }

// inline void print_spec(Qreal* f, int Nxh, int Ny){
//     for(int j = 0; j < Ny; j++){
//         for (int i = 0; i < Nxh; i++){
//             int index = i + j*Nxh;
//             printf("%.2f  ", f[index]);
//         }
//         cout << endl;
//     }
//     cout << endl;
// }

// __global__
// void vel_funcD(Qcomp *w_c, Qcomp *u_c, Qcomp *v_c, 
// Qreal* k_squared, Qreal* kx, Qreal*ky, int Nxh, int Ny, int BSZ){
//     int i = blockIdx.x * BSZ + threadIdx.x;
//     int j = blockIdx.y * BSZ + threadIdx.y;
//     int index = j*Nxh + i;
//     if (i<Nxh && j<Ny){
//         if (i==0 && j==0)
//         {
//             u_c[index] = make_hipDoubleComplex(0.0,0.0);
//             v_c[index] = make_hipDoubleComplex(0.0,0.0);
//         }
//         else{
//             //u = -D_y(\phi) -> u_spec = -1 * i* ky* w_spec/(-1* (kx^2+ky^2) )
//             u_c[index] = ky[index]*im()*w_c[index]/(k_squared[index]);
//             //v = D_x(\phi) -> v_spec = i* kx* w_spec/(-1* (kx^2+ky^2) )
//             v_c[index] = -1.0*kx[index]*im()*w_c[index]/(k_squared[index]);
//         }
//     }
// }
// inline void vel_func(Qcomp *w_c, Qcomp *u_c, Qcomp *v_c, Mesh *mesh){
//     vel_funcD<<<mesh->dimGridsp, mesh->dimBlocksp>>>(w_c, u_c, v_c, mesh->k_squared, mesh->kx, mesh->ky, mesh->Nxh, mesh->Ny, mesh->BSZ);
// }

// inline void coord(Qreal dx, Qreal dy, int Nx, int Ny){
//     ofstream xcoord("x.csv");
//     ofstream ycoord("y.csv");
//     for (int j=0; j<Ny; j++){
//         for ( int i=0; i< Nx; i++){
//             float x = dx*i;
//             float y = dy*j;
//             xcoord << x << ",";
//             ycoord << y << ",";
//         }
//         xcoord << endl;
//         ycoord << endl;
//     }
//     xcoord.close();
//     ycoord.close();
// }

int main(){
    int Nx = 8;
    int Ny = Nx;
    int BSZ = 16;
    int Nxh = Nx/2+1;
    int specsize = Nxh*Ny*sizeof(Qcomp);
    int physize = Nx*Ny*sizeof(Qreal);
    int wavesize = Nxh*Ny*sizeof(Qreal);
    Qreal Lx = 2*M_PI;
    Qreal Ly = Lx;
    Qreal dx = Lx/Nx;
    Qreal dy = Ly/Ny;

    hipfftHandle transf;
    hipfftHandle inv_transf;
    cufft_error_func( hipfftPlan2d( &(transf), Ny, Nx, HIPFFT_D2Z ) );
    cufft_error_func( hipfftPlan2d( &(inv_transf), Ny, Nx, HIPFFT_Z2D ) );

    dim3 dimGridp = dim3(int((Nx-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
    dim3 dimBlockp = dim3(BSZ, BSZ);

    dim3 dimGridsp = dim3(int((Nxh-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
    dim3 dimBlocksp = dim3(BSZ, BSZ);

    Mesh *mesh = new Mesh(Nx, Ny, Lx, Ly, BSZ);
    coord(dx, dy, Nx, Ny);

    Qreal *w;
    Qcomp *w_c, *u_c, *v_c;
    Qreal *u, *v;
    Qreal *kx, *ky, *k_squared;

    Qreal *p1, *p2, *p3;
    Qcomp *sp1, *sp2, *sp3;
    hipMalloc((void**)&w, sizeof(Qreal)*Nx*Ny);
    hipMalloc((void**)&u, sizeof(Qreal)*Nx*Ny);
    hipMalloc((void**)&v, sizeof(Qreal)*Nx*Ny);

    hipMalloc((void**)&w_c, sizeof(Qcomp)*Nxh*Ny);
    hipMalloc((void**)&u_c, sizeof(Qcomp)*Nxh*Ny);
    hipMalloc((void**)&v_c, sizeof(Qcomp)*Nxh*Ny);

    hipMalloc((void**)&kx, sizeof(Qreal)*Nxh*Ny);
    hipMalloc((void**)&ky, sizeof(Qreal)*Nxh*Ny);
    hipMalloc((void**)&k_squared, sizeof(Qreal)*Nxh*Ny);

    p1 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
    p2 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
    p3 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
    sp1 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny); 
    sp2 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny);
    sp3 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny);

    Qreal *kxh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
    Qreal *kyh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
    Qreal *ksh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
    
    
    winit(p1, Nx, Ny, dx, dy);
    hipMemcpy(w, p1, sizeof(Qreal)*Nx*Ny, hipMemcpyHostToDevice);
    FwdTrans(w_c, w, mesh);
    // cufft_error_func( hipfftExecD2Z(transf, w, w_c));
    hipMemcpy(p2, w, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
    field_visual(p2, "w0.csv", Nx, Ny);

    hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    cuda_error_func(hipDeviceSynchronize());
    cout << "w_c" << endl;
    print_spec(sp2, Nxh, Ny);
    cuda_error_func(hipDeviceSynchronize());
    
    // cuda_error_func(hipDeviceSynchronize());

    // BwdTrans(w, w_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
    BwdTrans(w, w_c, mesh);
    // cufft_error_func( hipfftExecZ2D(inv_transf, w_c, w));
    // hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    // cuda_error_func(hipDeviceSynchronize());
    // cout << "w_c 1" << endl;
    // print_spec(sp2, Nxh, Ny);
    // cuda_error_func(hipDeviceSynchronize());
    // coeff<<<dimGridp, dimBlockp>>>(w, Nx, Ny, BSZ);
    // hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    // cuda_error_func(hipDeviceSynchronize());
    // cout << "w_c 2" << endl;
    // print_spec(sp2, Nxh, Ny);
    hipMemcpy(p2, w, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
    field_visual(p2, "w.csv", Nx, Ny);

    cufft_error_func( hipfftExecD2Z(transf, w, w_c));
    hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    cuda_error_func(hipDeviceSynchronize());
    cout << "w_c 3" << endl;
    print_spec(sp2, Nxh, Ny);
    cuda_error_func(hipDeviceSynchronize());

    waveinit(kxh, kyh, ksh, Nxh, Ny, Lx, Ly);
    hipMemcpy(kx, kxh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);
    hipMemcpy(ky, kyh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);
    hipMemcpy(k_squared, ksh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);

    hipMemcpy(kxh, mesh->kx, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
    cout << "kx" << endl;
    print_spec(kxh,Nxh,Ny);
    hipMemcpy(kyh, mesh->ky, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
    cout << "ky" << endl;
    print_spec(kyh,Nxh,Ny);
    hipMemcpy(ksh, mesh->k_squared, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
    cout << "k_squared" << endl;
    print_spec(ksh,Nxh,Ny);

    cuda_error_func(hipDeviceSynchronize());
    // vel_func<<<dimGridsp, dimBlocksp>>>(w_c, u_c, v_c, mesh->k_squared, mesh->kx, mesh->ky, mesh->Nxh, mesh->Ny, mesh->BSZ);
    vel_func(w_c, u_c, v_c, mesh);
    hipMemcpy(sp2, u_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    cuda_error_func(hipDeviceSynchronize());
    cout << "u_c" << endl;
    print_spec(sp2, Nxh, Ny);
    hipMemcpy(sp2, v_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
    cuda_error_func(hipDeviceSynchronize());
    cout << "v_c" << endl;
    print_spec(sp2,Nxh, Ny);
    cuda_error_func(hipDeviceSynchronize());
    // BwdTrans(u, u_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
    BwdTrans(u,u_c,mesh);
    // cufft_error_func( hipfftExecZ2D(inv_transf, u_c, u));
    // coeff<<<dimGridp, dimBlockp>>>(u, Nx, Ny, BSZ);
    // BwdTrans(v, v_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
    BwdTrans(v,v_c,mesh);
    // cufft_error_func( hipfftExecZ2D(inv_transf, v_c, v));
    // coeff<<<dimGridp, dimBlockp>>>(v, Nx, Ny, BSZ);
    
    hipMemcpy(p1, u, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
    field_visual(p1, "u.csv", Nx, Ny);
    hipMemcpy(p2, v, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
    field_visual(p2, "v.csv", Nx, Ny);

    delete mesh;
    return 0;

}
// int main(){
//     int Nx = 8;
//     int Ny = Nx;
//     int BSZ = 16;
//     int Nxh = Nx/2+1;
//     int specsize = Nxh*Ny*sizeof(Qcomp);
//     int physize = Nx*Ny*sizeof(Qreal);
//     int wavesize = Nxh*Ny*sizeof(Qreal);
//     Qreal Lx = 2*M_PI;
//     Qreal Ly = Lx;
//     Qreal dx = Lx/Nx;
//     Qreal dy = Ly/Ny;

//     hipfftHandle transf;
//     hipfftHandle inv_transf;
//     cufft_error_func( hipfftPlan2d( &(transf), Ny, Nx, HIPFFT_D2Z ) );
//     cufft_error_func( hipfftPlan2d( &(inv_transf), Ny, Nx, HIPFFT_Z2D ) );

//     dim3 dimGridp = dim3(int((Nx-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
//     dim3 dimBlockp = dim3(BSZ, BSZ);

//     dim3 dimGridsp = dim3(int((Nxh-0.5)/BSZ) + 1, int((Ny-0.5)/BSZ) + 1);
//     dim3 dimBlocksp = dim3(BSZ, BSZ);


//     // define parameters
//     Qreal Rf = 0.0000075;
//     Qreal lambda = 0.1;
//     Qreal Re = 0.1;
//     Qreal Er = 0.1;
//     coord(dx, dy, Nx, Ny);

//     Qreal *w;
//     Qcomp *w_c, *u_c, *v_c;
//     Qreal *u, *v;
//     Qreal *kx, *ky, *k_squared;

//     Qreal *p1, *p2, *p3;
//     Qcomp *sp1, *sp2, *sp3;
//     hipMalloc((void**)&w, sizeof(Qreal)*Nx*Ny);
//     hipMalloc((void**)&u, sizeof(Qreal)*Nx*Ny);
//     hipMalloc((void**)&v, sizeof(Qreal)*Nx*Ny);

//     hipMalloc((void**)&w_c, sizeof(Qcomp)*Nxh*Ny);
//     hipMalloc((void**)&u_c, sizeof(Qcomp)*Nxh*Ny);
//     hipMalloc((void**)&v_c, sizeof(Qcomp)*Nxh*Ny);

//     hipMalloc((void**)&kx, sizeof(Qreal)*Nxh*Ny);
//     hipMalloc((void**)&ky, sizeof(Qreal)*Nxh*Ny);
//     hipMalloc((void**)&k_squared, sizeof(Qreal)*Nxh*Ny);

//     p1 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
//     p2 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
//     p3 = (Qreal*)malloc(sizeof(Qreal)*Nx*Ny);
//     sp1 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny); 
//     sp2 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny);
//     sp3 = (Qcomp*)malloc(sizeof(Qcomp)*Nxh*Ny);

//     Qreal *kxh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
//     Qreal *kyh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
//     Qreal *ksh = (Qreal*)malloc(sizeof(Qreal)*Nxh*Ny);
    
    
//     winit(p1, Nx, Ny, dx, dy);
//     hipMemcpy(w, p1, sizeof(Qreal)*Nx*Ny, hipMemcpyHostToDevice);
//     // FwdTrans(w_c, w, transf);
//     cufft_error_func( hipfftExecD2Z(transf, w, w_c));
//     hipMemcpy(p2, w, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
//     field_visual(p2, "w0.csv", Nx, Ny);
//     hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cuda_error_func(hipDeviceSynchronize());
//     cout << "w_c" << endl;
//     print_spec(sp2, Nxh, Ny);
//     cuda_error_func(hipDeviceSynchronize());
    
//     // cuda_error_func(hipDeviceSynchronize());

//     // BwdTrans(w, w_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
//     cufft_error_func( hipfftExecZ2D(inv_transf, w_c, w));
//     // hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     // cuda_error_func(hipDeviceSynchronize());
//     // cout << "w_c 1" << endl;
//     // print_spec(sp2, Nxh, Ny);
//     // cuda_error_func(hipDeviceSynchronize());
//     // coeff<<<dimGridp, dimBlockp>>>(w, Nx, Ny, BSZ);
//     // hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     // cuda_error_func(hipDeviceSynchronize());
//     // cout << "w_c 2" << endl;
//     // print_spec(sp2, Nxh, Ny);
//     hipMemcpy(p2, w, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
//     field_visual(p2, "w.csv", Nx, Ny);

//     cufft_error_func( hipfftExecD2Z(transf, w, w_c));
//     hipMemcpy(sp2, w_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cuda_error_func(hipDeviceSynchronize());
//     cout << "w_c 3" << endl;
//     print_spec(sp2, Nxh, Ny);
//     cuda_error_func(hipDeviceSynchronize());

//     waveinit(kxh, kyh, ksh, Nxh, Ny, Lx, Ly);
//     hipMemcpy(kx, kxh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);
//     hipMemcpy(ky, kyh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);
//     hipMemcpy(k_squared, ksh, sizeof(Qreal)*Nxh*Ny, hipMemcpyHostToDevice);

//     hipMemcpy(kxh, kx, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cout << "kxh" << endl;
//     print_spec(kxh,Nxh,Ny);
//     hipMemcpy(kyh, ky, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cout << "kyh" << endl;
//     print_spec(kyh,Nxh,Ny);
//     hipMemcpy(ksh, k_squared, sizeof(Qreal)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cout << "ksh" << endl;
//     print_spec(ksh,Nxh,Ny);

//     cuda_error_func(hipDeviceSynchronize());
//     vel_func<<<dimGridsp, dimBlocksp>>>(w_c, u_c, v_c, k_squared, kx, ky, Nxh, Ny, BSZ);
//     hipMemcpy(sp2, u_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cuda_error_func(hipDeviceSynchronize());
//     cout << "u_c" << endl;
//     print_spec(sp2, Nxh, Ny);
//     hipMemcpy(sp2, v_c, sizeof(Qcomp)*Nxh*Ny, hipMemcpyDeviceToHost);
//     cuda_error_func(hipDeviceSynchronize());
//     cout << "v_c" << endl;
//     print_spec(sp2,Nxh, Ny);
//     cuda_error_func(hipDeviceSynchronize());
//     // BwdTrans(u, u_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
//     cufft_error_func( hipfftExecZ2D(inv_transf, u_c, u));
//     coeff<<<dimGridp, dimBlockp>>>(u, Nx, Ny, BSZ);
//     // BwdTrans(v, v_c, Nx, Ny, BSZ, dimGridp, dimBlockp, inv_transf);
//     cufft_error_func( hipfftExecZ2D(inv_transf, v_c, v));
//     coeff<<<dimGridp, dimBlockp>>>(v, Nx, Ny, BSZ);
    
//     hipMemcpy(p1, u, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
//     field_visual(p1, "u.csv", Nx, Ny);
//     hipMemcpy(p2, v, sizeof(Qreal)*Nx*Ny, hipMemcpyDeviceToHost);
//     field_visual(p2, "v.csv", Nx, Ny);

//     return 0;
// }